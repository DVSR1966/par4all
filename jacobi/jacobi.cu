/* Adapted fro Terapix PIPS output */


#define P4A_DEBUG
#define P4A_ACCEL_CUDA
#include <p4a_accel.h>

#include <stdio.h>
#include <stdlib.h>
typedef float float_t;
#define SIZE 64
#define T 64

float_t space[SIZE][SIZE];
// For the dataparallel semantics:
float_t save[SIZE][SIZE];

void get_data(char filename[]) {
  int i, j, nx, ny;
  unsigned char c;
  FILE *fp;

  if ((fp = fopen(filename, "r")) == NULL) {
    perror("Error loading file");
    exit(0);
  }

  /* Get *.pgm file type */
  c = fgetc(fp);
  c = fgetc(fp);

  /* Skip comment lines */
  do {
    while((c = fgetc(fp)) != '\n');
  } while((c = fgetc(fp)) == '#');

  /* Put back good char */
  ungetc(c,fp);

  /* Get image dimensions */
  fscanf(fp, "%d %d\n", &nx, &ny);
  /* Get grey levels */
  fscanf(fp,"%d",&i);
  /* Get ONE carriage return */
  fgetc(fp);
  printf("Input image  : x=%d y=%d grey=%d\n", nx, ny, i);


  for(i = 0;i < SIZE; i++)
    for(j = 0;j < SIZE; j++) {
      c = fgetc(fp);
      space[i][j] = c;
    }

  fclose(fp);
}


void write_data(char filename[]) {
  int i,j;
  unsigned char c;
  FILE *fp;

  if ((fp = fopen(filename, "w")) == NULL) {
    perror("Error opening file");
    exit(0);
  }

  /* Write the PGM header: */
  fprintf(fp,"P5\n%d %d\n255\n", SIZE, SIZE);

  for(i = 0;i < SIZE; i++)
    for(j = 0;j < SIZE; j++) {
      c = save[i][j];
      fputc(c, fp);
    }
  fclose(fp);
}


#define MIN(a,b) (a < b ? a : b )
/*
 * file for kernel1.c
 */

P4A_ACCEL_KERNEL void kernel1(float_t save[64][64], float_t space[64][64], int i, int j)
{
   //int j;
   {
      //int i_1;
      // No need of strip mining in CUDA
      // for(i_1 = i; i_1 <= MIN(i+9, 62); i_1 += 1)
      // Already 2D
      //   for(j = 1; j <= 62; j += 1)
     save[i][j] = 0.25*(space[i-1][j]+space[i+1][j]+space[i][j-1]+space[i][j+1]);
   }
}


/*
 * file for launch_kernel1.c
 */
P4A_ACCEL_KERNEL_WRAPPER void kernel1_wrapper(float_t save[64][64], float_t space[64][64])
{
  int j;
  int i;
   /* Use 2 array in flip-flop to have dataparallel forall semantics. I
           could use also a flip-flop dimension instead... */
kernel1:
   //for(i = 1; i <= 62; i += 10)
  /* We need this wrapper to get the virtual processor coordinates

     The Cuda compiler inline P4A_ACCEL_KERNEL functions by default, so
     there is no overhead */
  i = P4A_VP_X;
  j = P4A_VP_Y;
  // Oops. I forgotten a loop normalize since the GPU iterate in [0..SIZE-1]...
  /* We need a phase to generate this clamping too: */
  if (i >= 1 && i <= 62 && j >= 1 && j <= 62)
    kernel1(save, space, i, j);
}

/*
 * file for launch_kernel1.c
 */
void launch_kernel1(float_t save[64][64], float_t space[64][64])
{
  // int j;
  // int i;
   /* Use 2 array in flip-flop to have dataparallel forall semantics. I
           could use also a flip-flop dimension instead... */
kernel1:
   //for(i = 1; i <= 62; i += 10)
  // Oops. I forgotten a loop normalize since the GPU iterate in [0..SIZE-1]...
  P4A_CALL_ACCEL_KERNEL_2D(kernel1_wrapper, SIZE, SIZE, save, space);
}

void compute(float_t save[64][64], float_t space[64][64]) {
  int i, j;

  /* Use 2 array in flip-flop to have dataparallel forall semantics. I
     could use also a flip-flop dimension instead... */
kernel1:   launch_kernel1(save, space);

  // The same should be done on this kernel2...
#pragma omp parallel for private(j)
   for(i = 1; i <= 62; i += 1)
#pragma omp parallel for 
      for(j = 1; j <= 62; j += 1)

         space[i][j] = 0.25*(save[i-1][j]+save[i+1][j]+save[i][j-1]+save[i][j+1]);
}


int main(int argc, char *argv[]) {
  int t;

  P4A_INIT_ACCEL;

  if (argc != 2) {
    fprintf(stderr,
	    "%s needs only one argument that is the PGM image input file\n",
	    argv[0]);
    exit(0);
  }
  get_data(argv[1]);

  /* Useless to transfer and allocate data everytime... So this should be
     put at the highest level. It needs the interprocedural PIPS region
     analysis... :-) */
  float_t (*p4a_var_space)[SIZE][SIZE];
  P4A_ACCEL_MALLOC(&p4a_var_space, sizeof(space));
  P4A_COPY_TO_ACCEL(space, p4a_var_space, sizeof(space));

  float_t (*p4a_var_save)[SIZE][SIZE];
  P4A_ACCEL_MALLOC(&p4a_var_save, sizeof(save));

  P4A_ACCEL_TIMER_START;

  for(t = 0; t < T; t++)
    compute(*p4a_var_space, *p4a_var_save);

  double execution_time = P4A_ACCEL_TIMER_STOP_AND_FLOAT_MEASURE();
  fprintf(stderr, "Temps d'ex�cution : %f ms\n", execution_time);

  P4A_COPY_FROM_ACCEL(save, p4a_var_save, sizeof(save));

  P4A_ACCEL_FREE(p4a_var_space);
  P4A_ACCEL_FREE(p4a_var_save);

  write_data("output.pgm");

  P4A_RELEASE_ACCEL;
  return 0;
}
